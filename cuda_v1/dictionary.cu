#include "hip/hip_runtime.h"
#include "dictionary.h"

__constant__ password seq[20];
__constant__ uint8_t target_hash[16];

void init_dictionary_seq(){
    cpu_seq = (password *)malloc(20*sizeof(password));

    memcpy(cpu_seq[0].word,"123", 3); cpu_seq[0].length = 3;
    memcpy(cpu_seq[1].word,"1234", 4); cpu_seq[1].length = 4;
    memcpy(cpu_seq[2].word,"12345", 5);   cpu_seq[2].length = 5;
    memcpy(cpu_seq[3].word,"123456", 6);   cpu_seq[3].length = 6;
    memcpy(cpu_seq[4].word,"1234567", 7);   cpu_seq[4].length = 7;
    memcpy(cpu_seq[5].word,"12345678", 8);   cpu_seq[5].length = 8;
    memcpy(cpu_seq[6].word,"123456789", 9);   cpu_seq[6].length = 9;
    memcpy(cpu_seq[7].word,"1234567890", 10);   cpu_seq[7].length = 10;
    memcpy(cpu_seq[8].word,"696969", 6);   cpu_seq[8].length = 6;
    memcpy(cpu_seq[9].word,"111111", 6);   cpu_seq[9].length = 6;
    memcpy(cpu_seq[10].word,"1111", 4);   cpu_seq[10].length = 4;
    memcpy(cpu_seq[11].word,"1212", 4);   cpu_seq[11].length = 4;
    memcpy(cpu_seq[12].word,"7777", 4);   cpu_seq[12].length = 4;
    memcpy(cpu_seq[13].word,"1004", 4);   cpu_seq[13].length = 4;
    memcpy(cpu_seq[14].word,"2000", 4);   cpu_seq[14].length = 4;
    memcpy(cpu_seq[15].word,"4444", 4);   cpu_seq[15].length = 4;
    memcpy(cpu_seq[16].word,"2222", 4);   cpu_seq[16].length = 4;
    memcpy(cpu_seq[17].word,"6969", 4);   cpu_seq[17].length = 4;
    memcpy(cpu_seq[18].word,"9999", 4);   cpu_seq[18].length = 4;
    memcpy(cpu_seq[19].word, "3333", 4);   cpu_seq[19].length = 4;

    hipMemcpyToSymbol(HIP_SYMBOL(seq), seq_cpu, sizeof(password)*20);  
    free(cpu_seq);
}

void init_target_hash_dictionary(uint8_t* int_test){
    hipMemcpyToSymbol(HIP_SYMBOL(target_hash), int_test, sizeof(uint8_t)*16);  
}

__global__ void mutate_and_check(password *dict, unsigned int numwords, int mutation_method, int* found){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= numwords) return;

    // First, try to copy the password from memory
    password new_pas;
    new_pas.length = dict[index].length;
    for(int i = 0; i < new_pas.length; ++i){
        new_pas.word[i] = dict[index].word[i];
    }

    // Then, try to mutate the input password
    if (mutation_method==0){
        /* First letter uppercase */
        if (new_pas.word[0] >= 'a' && new_pas.word[0] <= 'z')
            new_pas.word[0] +='A'-'a';
    }
    else if (mutation_method==1){
        /* Last letter uppercase */
        size_t len = new_pas.length;
        if (new_pas.word[len-1] >= 'a' && new_pas.word[len-1] <= 'z')
            new_pas.word[len-1] += 'A'-'a';
    }
    else if (mutation_method>=2 && mutation_method<=11){
        /* Add one digit to end
         * iterator: z-2    */
        size_t len = new_pas.length;
        new_pas.word[len] = '0' + z-2;
        new_pas.length += 1;
    }
   /* Add sequence of numbers at end; e.g. 1234, 84, 1999 */
    else  if (mutation_method>=12 && mutation_method<=111){
        // 0 to 99
        // iterator: z-12
        size_t len = new_pas.length;
        new_pas.word[len] = '0' + ((z-12)/10)%10;
        new_pas.word[len+1] = '0' + (z-12)%10;
        new_pas.length += 2;
    }
    else if (mutation_method>=112 && mutation_method<=231){
        // 1900 to 2020
        // iterator: z + (1900-112)
        size_t len = new_pas.length;
        new_pas.word[len] = '0' + ((z+1900-112)/1000)%10;
        new_pas.word[len+1] = '0' + ((z+1900-112)/100)%10;
        new_pas.word[len+2] = '0' + ((z+1900-112)/10)%10;
        new_pas.word[len+3] = '0' + (z+1900-112)%10;
        new_pas.length += 4;
    }
    else if (mutation_method>=232 && mutation_method<=251){
        // Other common sequences
        // iterator: z-232
        //sprintf(&temp,"%s",sequences[z-252]);
        size_t len = new_pas.length;
        for(int i = 0; i < seq[z-232].length; ++i){
            new_pas.word[len] + i = seq[z-232].word[i];
        }
        new_pas.length = len + seq[z-232].length;
    }

    word16 md5_hash;
    md5(&new_pas, (uint8_t*)md5_hash.word);
    int flag = 1;
    unsigned int j=0;
    while(j<16){
        if(target_hash[j] != (uint8_t)md5_hash.word[j]){
            flag = 0;
            break;
        }
        ++j;
    }
    if(flag == 1){
        atomicAdd(found, 1);
        printf("\n!!!!PASSWORD FOUND!!!!\nPassword is: ");
        for(int i = 0; i < new_pas.length; ++i){
            printf("%c", new_pas.word[i]);
        }
        printf("\n");
    }
}