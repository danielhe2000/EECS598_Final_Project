#include "hip/hip_runtime.h"
#include "brute_force.h"
#include "dictionary.h"
#include "support.h"
#include "md5.h"
#include <math.h>

void readPwdFromFile(FILE *infile, password **pwd, unsigned int *numLines){

	unsigned int numberOfLines = 0;
	int ch;
	while (EOF != (ch=getc(infile))){
    	if (ch=='\n'){
    		++numberOfLines;
	    	if(numberOfLines == (UINT_MAX/sizeof(char*)))
	    		break;
    	}
    }
    rewind(infile);

    *pwd = (password*)malloc(numberOfLines*sizeof(password));
    memset(*pwd, 0, numberOfLines*sizeof(password));
    if(*pwd == NULL){
        printf("\nERROR: Memory allocation did not complete successfully! Exiting.");
        exit(0);
    }

    char *line = NULL;
	size_t len = 0;
	int read_len = 0;
	unsigned int i=0;
	unsigned int toReduce = 0;
	while (i<numberOfLines) {
		read_len = getline(&line, &len, infile);
		if(read_len != -1){
			if(line[read_len-1] == '\n')    read_len = read_len - 1;
			if(line[read_len-1] == '\r')    read_len = read_len - 1;

			if(read_len > 45){
                //printf("Skipping (too big) - %s\n",line);
                ++toReduce;
            } else {
                // (*pwd)[i-toReduce] = (char*)malloc( (read_len+1)*sizeof(char));
                memcpy((*pwd)[i-toReduce].word,line,read_len);
                (*pwd)[i-toReduce].length = read_len;
                //printf("Pwd Read: %s, %d\n", (*pwd)[i], read_len);
	  		}
	  	} else {
            ++toReduce;
	  	}
        free(line);
        line = NULL;
		len = 0;
	  	i++;
	}
	*numLines = numberOfLines-toReduce;
	//passwd = &pwd;
}

inline void printpwd(password *pwd){
    unsigned int i=0;
    char *str = pwd->word;
    while(i < pwd->length) {
        printf("%c",str[i]);
		++i;
	}
}


void printall(password *pwd, unsigned int num){
	unsigned int i=0;
	while(i<num) {
		//char *str = pwd[i];
		printf("Pwd as Stored: ");
        printpwd(&(pwd[i]));
		printf("\n");
		++i;
	}
	printf("Num of lines : %d\n",num);
}

void hashToUint8(char *charHash, uint8_t intHash[]){
    char tempChar[16][3];
    int j=0;
    while(j<16){
        tempChar[j][0] = charHash[j*2];
        tempChar[j][1] = charHash[j*2+1];
        tempChar[j][2] = '\0';
        ++j;
    }
    j = 0;
    while(j<16){
        sscanf(tempChar[j], "%x", (unsigned int*)(&(intHash[j])));
        ++j;
    }
}

int main(int argc, char **argv){
    // to be tested
    char *test;
    uint8_t int_test[16];
    if (argc < 2) {
        printf("usage: %s 'stringhash'\n", argv[0]);
        return 1;
    }

    test = argv[1];
    if(strlen(test) != 32){
        printf ("Invalid hash. Exiting.\n");
		exit(0);
    }

    hashToUint8(test,int_test);

    // Initiate for 
    init_dictionary_seq();
    init_md5_const();
    init_target_hash_dictionary(int_test);
    init_target_hash_brute_force(int_test);

    const char *filename = "plaintext/mostcommon-10k";

    FILE *infile;
    if ((infile = fopen (filename, "r")) == NULL){
		printf ("%s can't be opened\n", filename);
		exit(0);
	}

    Timer totaltimer,filereadtimer, devicealloctime, MD5time, Dicttime;
    startTime(&totaltimer);

    startTime(&filereadtimer);
    unsigned int num_pwd;
    password *pwd;
    readPwdFromFile(infile, &pwd, &num_pwd);
    printf("Total Dictionary Words: %d\n",num_pwd);
    //printall(pwd, numPwd);
    stopTime(&filereadtimer);
    printf("File read time: %f s\n", elapsedTime(filereadtimer));

    startTime(&devicealloctime);
    password *device_password_array;
    hipMalloc((void**)&device_password_array, num_pwd*sizeof(password));
    hipMemcpy(device_password_array, pwd, num_pwd*sizeof(password), hipMemcpyHostToDevice); // stream by dma
    stopTime(&devicealloctime);
    printf("Device Allocation time: %f s\n", elapsedTime(devicealloctime));

    startTime(&MD5time);
    startTime(&Dicttime);
    int found = 0;

    // First we start dictionary
    for(int i = -1; found == 0 && i <= 251; ++i){
        // Copy found to cuda
        int* cuda_found;
        hipMalloc((void**)&cuda_found, sizeof(int));
        hipMemcpy(cuda_found, &found, sizeof(int), hipMemcpyHostToDevice);

        // Allocate size
        dim3 DimBlock(BLOCKTHREADS,1,1);
        dim3 DimGrid(num_pwd/BLOCKTHREADS, 1, 1);
        if(num_pwd%BLOCKTHREADS) DimGrid.x ++;

        // Run kernel
        uint *pwd_data = (uint *) device_password_array;
        mutate_and_check<<<DimGrid, DimBlock>>>(pwd_data, num_pwd, i, cuda_found);
        // printf("Possible error: %s\n", hipGetErrorString(hipGetLastError()));
        hipDeviceSynchronize();

        hipMemcpy(&found, cuda_found, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(cuda_found);
    }
    stopTime(&Dicttime);
    printf("Dictionary Calculation Time: %f s\n", elapsedTime(Dicttime));

    if(!found){
        printf("Couldn't find the password with dictionary manipulation\n");
        for(unsigned int i = 1; !found && i <= 7; ++i){
            size_t search_space = pow(26, i);
            // printf(">>>>>>>>>>>>>>>>Search space %d<<<<<<<<<<<<<<<\n", search_space);
            // Copy found to cuda
            int* cuda_found;
            hipMalloc((void**)&cuda_found, sizeof(int));
            hipMemcpy(cuda_found, &found, sizeof(int), hipMemcpyHostToDevice);

            // Allocate size
            dim3 DimBlock(1024,1,1);
            dim3 DimGrid(search_space/1024, 1, 1);
            if(search_space%1024) DimGrid.x ++;
            
            // Run kernel
            brute_force<<<DimGrid, DimBlock>>>(i, cuda_found);
            // printf("Possible error: %s\n", hipGetErrorString(hipGetLastError()));
            hipDeviceSynchronize();

            hipMemcpy(&found, cuda_found, sizeof(int), hipMemcpyDeviceToHost);
            hipFree(cuda_found);
        }
    }
    if(!found){
        printf("Sorry. Couldn't find the password\n");
    }
    stopTime(&MD5time);
    printf("MD5 Calculation time: %f s\n", elapsedTime(MD5time));

    stopTime(&totaltimer);
    printf("Total Time: %f s\n", elapsedTime(totaltimer));

    // Clean up
	if(infile != NULL)	fclose (infile);
    hipFree(device_password_array);


    return 0;
}